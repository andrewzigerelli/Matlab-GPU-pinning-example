#include "mex.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

static int isPinned = 0;
static double *pinnedA = NULL;
static mwSize dim;
void cleanup(void) {
    hipFree(pinnedA);
    mexPrintf("Removing pinned matrix A.\n");
}
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {

    const mwSize *dims;
    const mwSize *bdims;
    mwSize ndim;
    double *A, *b, *d_b, *d_y;
    double *y; 

    double *alpha;
    alpha = (double *) malloc(sizeof(double));
    *alpha = 1;
    double *beta;
    beta = (double *) malloc(sizeof(double));
    *beta = 0;

    /* setup cublas */
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    stat = hipblasCreate(&handle);
    if ( stat != HIPBLAS_STATUS_SUCCESS ) {
        mexErrMsgIdAndTxt(" MATLAB:Atimesb:cudaFailure", 
                "cublas initialization failed");
    }

    /* input should be matrix A and vector b, if first run
        else, input may just be vector b */
    if(nrhs == 1 && isPinned == 0) {
       mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidInputs",
               "You must supply the matrix A for the first run, i.e. y=Atimesb(A,b)");
    }
    else if (nrhs < 1) {
        mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidNumInputs",
                "input is required");
    }
    else if (nrhs == 2 && isPinned == 1) {
        mexPrintf("Warning: A is already on GPU; ignoring this A input.\n");
    }

    if(!isPinned) {
        //pin square matrix to GPU, size dim * dim
        ndim = mxGetNumberOfDimensions(prhs[0]);
        dims = mxGetDimensions(prhs[0]);
        if( (dims[0] != dims[1]) || (ndim !=2) ){
            mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidDims", 
                    "A must be a square matrix");
        }
        dim = dims[0];
        A = mxGetPr(prhs[0]);
        hipMalloc((void **)&pinnedA, dim*dim*sizeof(double));
        /* we should make sure hipMalloc doesn't fail */
        hipMemcpy(pinnedA, A, dim*dim*sizeof(double), hipMemcpyHostToDevice);
        mexPrintf("Copied A to GPU.\n");

        //Get vector b and transfer to GPU, size dim * 1
        b = mxGetPr(prhs[1]);
        bdims = mxGetDimensions(prhs[1]);
        if (bdims[0] != dim) {
            mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidDim",
                    "b must have same leading dimension as A");
        }
        if (bdims[1] != 1) {
            mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidDim",
                    "b must second dimension equal to 1");
        }
        ndim = mxGetNumberOfDimensions(prhs[1]);
        if(ndim !=2) {
            mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidDims", 
                    "b must be a vector");
        }

        hipMalloc((void **)&d_b, dim*sizeof(double));
        hipMemcpy(d_b, b, dim*sizeof(double), hipMemcpyHostToDevice);
        mexPrintf("Copied b to GPU.\n");

        //Calculate y=A*b
        /* see Dgemv documentation for explanation of params */
        hipMalloc((void **)&d_y, dim*sizeof(double));
        mexPrintf("Before gemv!\n");
        stat = hipblasDgemv(handle, HIPBLAS_OP_N, 
                (int) dim, (int) dim, 
                alpha, 
                pinnedA, (int) dim,
                d_b, 1, 
                beta, 
                d_y, 1);
        mexPrintf("After gemv!\n");
        
        //Copy output
        plhs[0] = mxCreateDoubleMatrix(dim,1,mxREAL);
        y = mxGetPr(plhs[0]);
        mexPrintf("Copying result to y\n");
        hipMemcpy(y, d_y, dim*sizeof(double), hipMemcpyDeviceToHost);
        mexPrintf("Done copying result to y\n");
       
        //cleanup
        hipFree(d_b);
        hipFree(d_y);


        //set pinned flag and register mex cleanup callback
        mexAtExit(cleanup);
        isPinned =1;
    }
    else {
        //get b vector and check dimensions
        if(nrhs==2) {
            b = mxGetPr(prhs[1]);
            bdims = mxGetDimensions(prhs[1]);
            ndim = mxGetNumberOfDimensions(prhs[1]);
        }
        else {
            b = mxGetPr(prhs[0]);
            bdims = mxGetDimensions(prhs[0]);
            ndim = mxGetNumberOfDimensions(prhs[0]);
        }
        if (bdims[0] != dim) {
            mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidDim",
                    "b must have same leading dimension as A");
        }
        if (bdims[1] != 1) {
            mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidDim",
                    "b must second dimension equal to 1");
        }
        if(ndim !=2) {
            mexErrMsgIdAndTxt( "MATLAB:Atimesb:invalidDims", 
                    "b must be a vector");
        }
        hipMalloc((void **)&d_b, dim*sizeof(double));
        hipMemcpy(d_b, b, dim*sizeof(double), hipMemcpyHostToDevice);
        mexPrintf("Copied b to GPU.\n");

        //Calculate y=A*b
        /* see Dgemv documentation for explanation of params */
        hipMalloc((void **)&d_y, dim*sizeof(double));
        mexPrintf("Before gemv!\n");
        stat = hipblasDgemv(handle, HIPBLAS_OP_N, 
                (int) dim, (int) dim, 
                alpha, 
                pinnedA, (int) dim,
                d_b, 1, 
                beta, 
                d_y, 1);
        mexPrintf("After gemv!\n");
        
        //Copy output
        plhs[0] = mxCreateDoubleMatrix(dim,1,mxREAL);
        y = mxGetPr(plhs[0]);
        mexPrintf("Copying result to y\n");
        hipMemcpy(y, d_y, dim*sizeof(double), hipMemcpyDeviceToHost);
        mexPrintf("Done copying result to y\n");
    }
}
